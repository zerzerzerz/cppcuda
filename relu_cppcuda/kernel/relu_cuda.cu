#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

#define DIV_UP(n,x) ((n-1+x)) / x
const dim3 block(16,16);

template <typename scalar_t>
__global__ void relu_kernel_forward(
    // 32表示index是32位的
    // scalar_t是模板参数
    // 2是tensor有几个维度
    // torch::RestrictPtrTraits对应const __restrict__ *
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> output
){
    const size_t b = blockIdx.y * blockDim.y + threadIdx.y;
    const size_t f = blockIdx.x * blockDim.x + threadIdx.x;
    if((b < input.size(0)) && (f < input.size(1))){
        output[b][f] = (input[b][f]>0)? input[b][f]: scalar_t(0);
    }
    return;
}


torch::Tensor relu_cuda_forward(const torch::Tensor& input){
    auto output = torch::zeros_like(input);
    const dim3 grid(DIV_UP(input.size(1), block.x), DIV_UP(input.size(0), block.y));

    // 动态类型分配，根据输入的类型自动确定模板参数
    // 第一个参数是输入的类型
    // 第二个参数是error字符串
    // 第三个参数是调用的匿名函数(lambda表达式)
    AT_DISPATCH_FLOATING_TYPES(input.type(), "relu_cuda_forward", ([&] {
        // <scalar_t>告诉kernel模板参数，在AT_DISPATCH_FLOATING_TYPES下只能为这个名字！！！
        // 不需要写template <typename scalar_t>
        // 其实是个#define
        relu_kernel_forward<scalar_t><<<grid,block>>>(
            // 从tensor到accessor，注意这是个函数，要加()来调用
            input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
            output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
        );
    }));

    return output;
}



template <typename scalar_t>
__global__ void relu_kernel_backward(
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> grad_output,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> grad_input
){
    const size_t b = blockIdx.y * blockDim.y + threadIdx.y;
    const size_t f = blockIdx.x * blockDim.x + threadIdx.x;
    if((b<grad_output.size(0)) && (f<grad_output.size(1))){
        grad_input[b][f] = (output[b][f]>0)? grad_output[b][f]: scalar_t(0);
    }
    return;
}


torch::Tensor relu_cuda_backward(const torch::Tensor& grad_output, const torch::Tensor& output){
    auto grad_input = torch::zeros_like(grad_output);
    const dim3 grid(DIV_UP(output.size(1), block.x), DIV_UP(output.size(0), block.y));

    AT_DISPATCH_FLOATING_TYPES(grad_output.type(), "relu_cuda_backward", ([&]{
        relu_kernel_backward<scalar_t><<<grid, block>>>(
            grad_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
            output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
            grad_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
        );
    }));

    return grad_input;
}